#include <nvbench/nvbench.cuh>
#include <cxxopts.hpp>
#include <gunrock/algorithms/algorithms.hxx>
#include <gunrock/algorithms/pr.hxx>

#include "benchmarks.hxx"

using namespace gunrock;
using namespace memory;

using vertex_t = int;
using edge_t = int;
using weight_t = float;

std::string filename;

struct parameters_t {
  std::string filename;
  bool help = false;
  cxxopts::Options options;

  /**
   * @brief Construct a new parameters object and parse command line arguments.
   *
   * @param argc Number of command line arguments.
   * @param argv Command line arguments.
   */
  parameters_t(int argc, char** argv) : options(argv[0], "PR Benchmarking") {
    options.allow_unrecognised_options();
    // Add command line options
    options.add_options()("h,help", "Print help")  // help
        ("m,market", "Matrix file",
         cxxopts::value<std::string>());  // mtx

    // Parse command line arguments
    auto result = options.parse(argc, argv);

    if (result.count("help")) {
      help = true;
      std::cout << options.help({""});
      std::cout << "  [optional nvbench args]" << std::endl << std::endl;
      // Do not exit so we also print NVBench help.
    } else {
      if (result.count("market") == 1) {
        filename = result["market"].as<std::string>();
        if (!util::is_market(filename)) {
          std::cout << options.help({""});
          std::cout << "  [optional nvbench args]" << std::endl << std::endl;
          std::exit(0);
        }
      } else {
        std::cout << options.help({""});
        std::cout << "  [optional nvbench args]" << std::endl << std::endl;
        std::exit(0);
      }
    }
  }
};

void pr_bench(nvbench::state& state) {
  // --
  // Add metrics
  state.collect_dram_throughput();
  state.collect_l1_hit_rates();
  state.collect_l2_hit_rates();
  state.collect_loads_efficiency();
  state.collect_stores_efficiency();

  // --
  // Build graph + metadata
  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;
  auto [properties, coo] = mm.load(filename);

  format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t> csr;
  csr.from_coo(coo);

  // --
  // Build graph

  auto G = graph::build<memory_space_t::device>(properties, csr);

  // --
  // Params and memory allocation
  srand(time(NULL));

  weight_t alpha = 0.85;
  weight_t tol = 1e-6;

  vertex_t n_vertices = G.get_number_of_vertices();
  thrust::device_vector<weight_t> p(n_vertices);

  // --
  // Run PR with NVBench
  state.exec(nvbench::exec_tag::sync, [&](nvbench::launch& launch) {
    gunrock::pr::run(G, alpha, tol, 100, p.data().get());
  });
}

int main(int argc, char** argv) {
  parameters_t params(argc, argv);
  filename = params.filename;

  if (params.help) {
    // Print NVBench help.
    const char* args[1] = {"-h"};
    NVBENCH_MAIN_BODY(1, args);
  } else {
    // Remove all gunrock parameters and pass to nvbench.
    auto args = filtered_argv(argc, argv, "--market", "-m", filename);
    NVBENCH_BENCH(pr_bench);
    NVBENCH_MAIN_BODY(args.size(), args.data());
  }
}
